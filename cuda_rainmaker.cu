#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <vector>
#include <limits>
#include <iostream>
#include <unistd.h>
#define THREADS 512

#ifdef __cplusplus
extern "C"
{
#endif

using namespace std; 

float *cu_grid;

// initialize grid with all 0 values
float* cu_init_grid(int h, int w) {
    cu_grid = (float*) malloc(sizeof(float*)*h*w);
    for (int k = 0; k < h*w; k++) {
        cu_grid[k] = 0;
    }
    return cu_grid;
}

void cu_display(float* grid, int h, int w) {
    cout << "\033[2J\033[1;1H";
    for (int i = 0; i < h; i++) {
        printf("\n");
        for (int j = 0; j < w; j++) {
            if (grid[i*w+j] < 1) {
                printf("    ");
            } else if (grid[i*w+j] < 2) {
                printf(".   ");
            } else if (grid[i*w+j] < 3) {
                printf("..  ");
            } else {
                printf("... ");
            } 
        }
    }
}

// Grid's evolve function while considering previous points
__global__ void cu_next_iteration(float *grid1, float *grid2, float* row, int h, int w, int size)
{
    int start = blockIdx.x*w;
    if (start >= w*h) {
        return;
    }

    // int drizzle;
    if (start == 0) {
        for (int i=0; i < w; i++) {
            grid2[i] = row[i];
        }
    } else {
        for (int i=start; i < start + w; i++) {
            grid2[i] = 0;
        }
        for (int i=start; i < start + w; i++) {
            if (grid1[i-w] >= size) {
                // if (j-1 >= 0) {
                grid2[i] += grid1[i-w]/2;
                // }
                if (i+1 < start + w) {
                    grid2[i+1] += grid1[i-w]/2;
                }
                // empty[j] += grid[i][j]/3;
            } else {
                grid2[i] += grid1[i-w];
            }
        }
    }
}

void cuda_rainmaker(int show, int iter, int height, int width, int size_threshold, unsigned int microseconds)
{
    int cu_height=height;
    int cu_width=width;
    int cu_size_threshold = size_threshold;
    int grid_dim = cu_height*cu_width;
    cu_grid = cu_init_grid(cu_height, cu_width);
    float *row = (float *)malloc(sizeof(float)*cu_width);
    float *device_row;
    srand (time(NULL));
    struct timeval time_start, time_end;
    long int kernel_time=0;

    // cuda 
    float *device_grid1, *device_grid2, *tmp;
    // hipEvent_t event;
    // hipEventCreate(&event);
    hipMalloc((void **) &device_grid1, sizeof(float)*grid_dim);
    hipMalloc((void **) &device_grid2, sizeof(float)*grid_dim);
    hipMalloc((void **) &device_row, sizeof(float)*cu_width);
    hipMemcpy(device_grid1, cu_grid, sizeof(float)*grid_dim, hipMemcpyHostToDevice);
    // hipMemcpy(device_grid2, cu_grid, sizeof(float)*grid_dim, hipMemcpyHostToDevice);//
    dim3 dimGrid(cu_height);
    dim3 dimBlock(1);

    for (int i = 0; i < iter; i++) {
        // cu_display1(cu_grid, cu_height, cu_width);
        if (show == 1) {
            usleep(100000); 
            cu_display(cu_grid, cu_height, cu_width);
        }
        // printf("\n");
        if (iter - i < cu_height) {
            for (int j=0; j < cu_width; j++) {
                row[j] = 0;
            // printf("%.1f, ", row[j]);
            }
        } else {
            for (int j=0; j < cu_width; j++) {
                row[j] = rand() % 4;
            // printf("%.1f, ", row[j]);
            }
        }
        hipMemcpy(device_row, row, sizeof(float)*cu_width, hipMemcpyHostToDevice);
        gettimeofday(&time_start, NULL);
        cu_next_iteration<<<dimGrid, dimBlock>>>(device_grid1, device_grid2, device_row, cu_height, cu_width, cu_size_threshold);
        gettimeofday(&time_end, NULL);
        kernel_time += (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
        tmp = device_grid2;
        device_grid2 = device_grid1;
        device_grid1 = tmp;
        // cu_split_and_merge<<<dimGrid, dimBlock>>>(device_grid1, device_grid2, cu_height, cu_width, cu_size_threshold);
        // hipMemcpy(cu_grid, device_grid2, sizeof(float)*grid_dim, hipMemcpyDeviceToHost);  
        // hipMemcpy(device_grid1, cu_grid, sizeof(float)*grid_dim, hipMemcpyHostToDevice);
        // cu_split_and_merge<<<dimGrid, dimBlock>>>(device_grid1, device_grid2, cu_height, cu_width, cu_size_threshold);
        // hipMemcpy(device_grid2, cu_grid, sizeof(float)*grid_dim, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        hipMemcpy(cu_grid, device_grid1, sizeof(float)*grid_dim, hipMemcpyDeviceToHost);  
    }
    // hipDeviceSynchronize();
    // hipEventSynchronize(event);  
    if (show == 1) 
        cout << "\033[2J\033[1;1H"; 
    printf("kernel time: %ld microseconds\n", kernel_time);
    // free back to heap
    hipFree(device_grid1);
    hipFree(device_grid2);
    free(cu_grid);
}

#ifdef __cplusplus
}
#endif
